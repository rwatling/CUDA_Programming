#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include "nvmlClass.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements, int workThreads, int idleThreads) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i <= (workThreads - idleThreads)) {
    if (i < numElements) {
      C[i] = A[i] + B[i] + 0.0f;
    }
  }
}

/**
 * Host main routine
 */
int main(void) {

  /************************NVML get device********************************/
  int nvml_dev {};
  hipError_t cuda_err;
  hipGetDevice( &nvml_dev );
  cuda_err = hipSetDevice( nvml_dev );


  if (cuda_err != hipSuccess) {
    std::cerr << "hipSetDevice failed for nvml\n" << std::endl;
  }

  /*************************CUDA Timing***********************************/
  hipEvent_t start, stop;
  float milliseconds;
  int iterations = 2000000;
  int numIdle = 64;

  std::string nvml_filename = "./vectorAdd_idle64_r1.csv";
  std::vector<std::thread> cpu_threads;
  std::string type;

  type.append("idle64_r1_vectorAdd_compute");
  nvmlClass nvml( nvml_dev, nvml_filename, type);

  cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  nvml.log_start();

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  printf("[Vector addition of %d elements]\n", numElements);

  // Allocate the host input vector A
  float *h_A = (float *)malloc(size);

  // Allocate the host input vector B
  float *h_B = (float *)malloc(size);

  // Allocate the host output vector C
  float *h_C = (float *)malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  //printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);


   //Default:
   //Blocks: 196
   //Threads: 256

   nvml.log_point();

   //Timing
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   for (int i = 0; i < iterations; i++) {
     vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements, threadsPerBlock, numIdle);
     err = hipGetLastError();
   }

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  nvml.log_point();

  std::cout << "Kernel elapsed time: " << milliseconds << " (ms)" << std::endl << std::endl;

  //std::cout << "Total blocks: " << blocksPerGrid << std::endl;
  //std::cout << "Threads per block: " << threadsPerBlock << std::endl;

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify that the result vector is correct
  /*for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");*/

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  //printf("Done\n");

  nvml.log_stop();

  // NVML
  // Create thread to kill GPU stats
  // Join both threads to main
  cpu_threads.emplace_back(std::thread( &nvmlClass::killThread, &nvml));

  for (auto& th : cpu_threads) {
    th.join();
    th.~thread();
  }

  cpu_threads.clear();
  nvml_filename.clear();
  type.clear();

  return 0;
}
