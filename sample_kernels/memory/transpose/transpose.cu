#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <assert.h>

#include "cuda_includes.h"
#include "nvmlClass.h"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 1;

__device__ int getGlobalIdx_3D_3D() {

  int blockId = blockIdx.x + blockIdx.y * gridDim.x
  + gridDim.x * gridDim.y * blockIdx.z;

  int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
  + (threadIdx.z * (blockDim.x * blockDim.y))
  + (threadIdx.y * blockDim.x)  + threadIdx.x;

  return threadId;
}

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed) {
    printf("%25.2f", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
    printf("%25.4f\n", ms / NUM_REPS );
  }
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
/*__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}*/

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(float *odata, const float *idata, int workThreads, int idleThreads)
{
  //int my_id = getGlobalIdx_3D_3D();

  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;


  if (threadIdx.x <= (workThreads - idleThreads)) {
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];
  }

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  if (threadIdx.x <= (workThreads - idleThreads)) {
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}


// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded
// to avoid shared memory bank conflicts.
/*__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}*/

int main(int argc, char **argv) {
  //NVML Stuff
  int devId = 0;
  std::string nvml_filename = "./transpose_idle1024_r5.csv";
  std::vector<std::thread> cpu_threads;
  std::string type;

  int iterations = 400000;

  type.append("idle1024_r5_transpose_memory");
  nvmlClass nvml( devId, nvml_filename, type);

  cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  nvml.log_start();

  const int nx = 1024;
  const int ny = 1024;

  //const int TILE_DIM = 32;
  //const int BLOCK_ROWS = 8;
  //const int NUM_REPS = 1;

  //1024 blocks 256 threads

  const int mem_size = nx*ny*sizeof(float);

  dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
  int workThreads = (TILE_DIM * BLOCK_ROWS);
  int idleThreads = 64;

  //int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  /*printf("\nDevice : %s\n", prop.name);
  printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n",
         nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);*/

  checkCuda( hipSetDevice(devId) );

  float *h_idata = (float*)malloc(mem_size);
  float *h_cdata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);

  float *d_idata, *d_cdata, *d_tdata;
  checkCuda( hipMalloc(&d_idata, mem_size) );
  checkCuda( hipMalloc(&d_cdata, mem_size) );
  checkCuda( hipMalloc(&d_tdata, mem_size) );

  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }

  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_idata[j*nx + i] = j*nx + i;

  // correct result for error checking
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      gold[j*nx + i] = h_idata[i*nx + j];

  // device
  checkCuda( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  //Defaults:
  /*
  const int TILE_DIM = 32;
  const int BLOCK_ROWS = 8;
  const int NUM_REPS = 1;
  */

  // ------------------
  // transposeCoalesced
  // ------------------
  checkCuda( hipMemset(d_tdata, 0, mem_size) );

  nvml.log_point();

  checkCuda( hipEventRecord(startEvent, 0) );

  for (int i = 0; i < iterations; i++) {
    transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata, workThreads, idleThreads);
  }

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

  nvml.log_point();

  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );

  // ------------
  // time kernels
  // ------------
  //printf("%25s%25s%25s\n", "Routine", "Bandwidth (GB/s)", "Time (ms)");
  //printf("%25s", "coalesced transpose");
  //postprocess(gold, h_tdata, nx * ny, ms);

  printf("Time (ms): %.4f\n", ms);

  std::cout << "Total blocks: " << nx/TILE_DIM * ny/TILE_DIM << std::endl;
  std::cout << "Threads per block: " << TILE_DIM * BLOCK_ROWS << std::endl;

  nvml.log_stop();

  // NVML
  // Create thread to kill GPU stats
  // Join both threads to main
  cpu_threads.emplace_back(std::thread( &nvmlClass::killThread, &nvml));

  for (auto& th : cpu_threads) {
    th.join();
    th.~thread();
  }

  cpu_threads.clear();
  nvml_filename.clear();
  type.clear();


error_exit:
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipFree(d_tdata) );
  checkCuda( hipFree(d_cdata) );
  checkCuda( hipFree(d_idata) );
  free(h_idata);
  free(h_tdata);
  free(h_cdata);
  free(gold);
}
