#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "nvmlClass.h"

#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

template <typename T>
__global__ void offset(T* a, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  //int blockSize = 256;
  //float ms;

  T *d_a;
  //hipEvent_t startEvent, stopEvent;


  //Default, nMB = 4
  //Sp 4 * 1024 * 1024 / sizeof(double) = 4 * 1024 * 1024 / 8 = 524288
  int n = nMB*1024*1024/sizeof(T);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  /*checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );*/

  /*printf("Offset, Bandwidth (GB/s):\n");

  offset<<<n/blockSize, blockSize>>>(d_a, 0); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    offset<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  printf("\n");
  printf("Stride, Bandwidth (GB/s):\n");*/

  /************************NVML get device********************************/
  int nvml_dev {};
  hipError_t cuda_err;
  hipGetDevice( &nvml_dev );
  cuda_err = hipSetDevice( nvml_dev );

  /*************************CUDA Timing***********************************/
  hipEvent_t start, stop;
  float milliseconds;
  int iterations = 10000;

  // Original
  // n/blockSize = 4096 blocks
  // blockSize = 256 threads

  //Same ratio
  // blocks 2048, threads 512
  // blocks 1024, threads 1024

  // Change Blocks
  // blocks 2048 threads 256
  // blocks 1024 threads 256
  // blocks 512 threads 256

  // Change Threads
  // blocks 4096 threads 512
  // block 4096 threads 128
  // block 4096 threads 64

  if (cuda_err != hipSuccess) {
    std::cerr << "hipSetDevice failed for nvml\n" << std::endl;
  }

  std::string nvml_filename = "./coalescing_default.csv";
  std::vector<std::thread> cpu_threads;
  std::string type;

  type.append("coalescing_memory");
  nvmlClass nvml( nvml_dev, nvml_filename, type);

  cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  nvml.log_start();

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < iterations; i++) {
    //stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
    stride<<<4096, 256>>>(d_a, 1);
  }

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  nvml.log_stop();

  // NVML
  // Create thread to kill GPU stats
  // Join both threads to main
  cpu_threads.emplace_back(std::thread( &nvmlClass::killThread, &nvml));

  for (auto& th : cpu_threads) {
    th.join();
    th.~thread();
  }

  cpu_threads.clear();
  nvml_filename.clear();
  type.clear();

  std::cout << "Kernel elapsed time: " << milliseconds << " (ms)" << std::endl << std::endl;

  /*for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );*/
  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int nMB = 4;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }

  hipDeviceProp_t prop;

  checkCuda( hipSetDevice(deviceId) )
  ;
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);

  printf("%s Precision\n", bFp64 ? "Double" : "Single");

  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);
}
