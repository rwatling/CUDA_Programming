#include "hip/hip_runtime.h"
#include "rand_init.h"

__global__ void rand_init(int* all_arrays, int num_arrays, int size) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int maxRand = 10;
	hiprandState state;
	unsigned long long seed = clock();

	hiprand_init(seed + thread_id, 0, 0, &state);

	all_arrays[thread_id] = (int) (hiprand_uniform(&state) * maxRand);
}
