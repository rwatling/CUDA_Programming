#include "hip/hip_runtime.h"
#include "shfl_match.h"

__global__ void shfl_match(int* all_arrays, int* match_array, int num_arrays, int size, unsigned long long* elapsed) {

	// Essential variables
	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int lane_id = threadIdx.x % WARP_SIZE;
	int current_num;
	int prev_num;
	unsigned long long start;
	unsigned long long stop;

	//For verification
	int* current_array = all_arrays + (thread_id * size);

	//For lane0 to get lane31 data???
	//__shared__ int shared_arrays[WARP_SIZE];

	//For random number generation
	int maxRand = 100;
	hiprandState state;
	unsigned long long seed = clock();

	hiprand_init(seed + thread_id, 0, 0, &state);

	if (thread_id >= num_arrays) { return; }

	if (thread_id == 0) {
		start = clock();
	}

	int match = 0;

	for (int i = 0; i < size; i++) {
		current_num = (int) (hiprand_uniform(&state) * maxRand);
		current_array[i] = current_num; //for verification
		prev_num = __shfl_sync(0xffffffff, current_num, lane_id - 1);

		if (current_num == prev_num) {
			match = 1;
		}
	}

	match_array[thread_id] = match;

	__syncthreads();

	if (thread_id == 0) {
		stop = clock();
		*elapsed = stop - start;
	}
}
