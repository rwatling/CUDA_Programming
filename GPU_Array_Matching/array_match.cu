#include "hip/hip_runtime.h"
#include "array_match.h"

__global__ void array_match(int* all_arrays, int* match_array, int num_arrays,  int size) {
	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;

	//For random number generation
	int maxRand = 100;
	hiprandState state;
	unsigned long long seed = clock();

	hiprand_init(seed + thread_id, 0, 0, &state);

	if (thread_id >= num_arrays) { return; }

	int* current_array = all_arrays + (thread_id * size); //Pointer arithmetic
	int* prev_array = all_arrays + ((thread_id - 1) * size); //Pointer arithmetic
	int match = 0;

	if (thread_id > 0) {

		for (int i = 0; i < size; i++) {
				//At runtime moment, generate random number
				int rand_num = (int) (hiprand_uniform(&state) * maxRand);;
				current_array[i] = rand_num;		
		}
	} else if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			//At runtime moment, generate random number
			current_array[i] = (int) (hiprand_uniform(&state) * maxRand);
		}
	}

	__syncthreads();

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			if (current_array[i] == prev_array[j]) {
				match = 1;
				break;
			}
		}

		if (match) {
			match_array[thread_id] = 1;
			break;
		}
	}
}
