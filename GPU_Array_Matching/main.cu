/* This program searches "states" for matches in their arrays using CUDA
*
* Author: Robbie Watling
*/

#include "cuda_includes.h"
#include "array_match.h"
#include "shm_array_match.h"
#include <iostream>
#include <vector>
#include <utility>
#include <iostream>
#include <sys/time.h>

using namespace std;

int main(int argc, char** argv) {

	/***Variable Declarations***/
	int* host_arrays;
	int* device_arrays;
	int* host_match;
	int* device_match;

	int array_size;
	int match_size;
	int num_arrays;
	int NUM_THREADS;
	int NUM_BLOCKS;
	int shared;

	size_t one_t;
	size_t array_set_bytes;
	size_t match_bytes;

	hipError_t cuda_err;

	struct timeval startShm;
	struct timeval startG;
	struct timeval stopShm;
	struct timeval stopG;

	double elapsed;

	if (argc < 4) {
		cerr << "./main array_size num_arrays shared(y/n)" << endl;
		return -1;
	}

	/***Initialization***/
	array_size = atoi(argv[1]);
	num_arrays = atoi(argv[2]);
	shared = atoi(argv[3]);
	match_size = num_arrays;
	NUM_THREADS = num_arrays;
	NUM_BLOCKS = 1;

	// Host allocation
	one_t = (size_t) 1;
	array_set_bytes = (size_t) num_arrays * array_size * sizeof(int);
	match_bytes = (size_t) match_size * sizeof(int);

	host_arrays = (int*) calloc(one_t, array_set_bytes);
	host_match = (int*) calloc(one_t, match_bytes);


	if (host_arrays == NULL) {
		cerr << "Host arrays calloc failed\n" << endl;
		return -1;
	}

	if (host_match == NULL) {
		cerr << "Host match calloc failed\n" << endl;
		return -1;
	}

	//Device Allocation
	cuda_err = hipMalloc((void**)&device_arrays, array_set_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allocation for array set failed" << endl;
		return -1;
	}

	cuda_err = hipMalloc((void**)&device_match, match_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allcoation for match array failed" << endl;
		return -1;
	}

	hipMemset(device_arrays, 0, array_size);
	hipMemset(device_match, 0, match_size);

	//If shared is specified
	if (shared) {

		//Start timer shm
		gettimeofday(&startShm, 0);

		/*** Search arrays and copy result back to host using shared memory***/
		//get maximum size of shared memory I can use
		shm_array_match <<<NUM_BLOCKS, NUM_THREADS, num_arrays * array_size * sizeof(int) >>> (device_arrays, device_match, num_arrays, array_size);

		gettimeofday(&stopShm, 0);

		//Copy match back to host
		hipMemcpy(host_match, device_match, match_bytes, hipMemcpyDeviceToHost);

		//Copy gpu arrays to host for verification
		hipMemcpy(host_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

		long shm_sec = stopShm.tv_sec - startShm.tv_sec;
		long shm_ms = stopShm.tv_usec - startShm.tv_usec;
		elapsed = shm_sec + shm_ms*1e-6;
	}

	//If not shared is specified
	if (!shared) {

		gettimeofday(&startG, 0);

		/*** Search arrays and copy back to host using global memory ***/
		array_match <<<NUM_BLOCKS, NUM_THREADS >>> (device_arrays, device_match, num_arrays, array_size);

		gettimeofday(&stopG, 0);

		//Copy match back to host
		hipMemcpy(host_match, device_match, match_bytes, hipMemcpyDeviceToHost);

		//Copy gpu arrays to host for verification
		hipMemcpy(host_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

		long g_sec = stopG.tv_sec - startG.tv_sec;
		long g_ms = stopG.tv_usec - startG.tv_usec;
		elapsed = g_sec + g_ms*1e-6;
	}

	cout << shared << "," << num_arrays << "," << array_size << "," << elapsed << endl;

	/*** Check arrays ***/
	int* temp_match = (int*) calloc(one_t, match_bytes);

	if (temp_match == NULL) {
		cerr << "Temp match allocation failed" << endl;
		return -1;
	}

	int bool_match;
	for (int i = 1; i < num_arrays; i++) {
		int step = i * array_size;
		int step2 = (i-1) * array_size;

		bool_match = 0;

		for (int k = 0; k < array_size; k++) {
			int i_element = host_arrays[step + k];

			for (int l = 0; l < array_size; l++) {
				int j_element = host_arrays[step2 + l];

				if (i_element == j_element) {
					temp_match[i] = 1;
					bool_match = 1;
					break;
				}

			}

			if (bool_match) {
				break;
			}
		}
	}

	//verify match arrays
	for (int i = 0; i < match_size; i++) {
		if(host_match[i] != temp_match[i]) {
			cerr << "Incorrect answer" << endl;
			cerr << "host_match[i]: " << host_match[i] << " at index " << i << endl;
			cerr << "temp_match[i]: " << temp_match[i] << endl;
			
			cerr << "all arrays: " << endl;
			for (int j = 0; j < num_arrays; j++) {
				int step = j * array_size;

				cerr << "[ ";
				for (int k = 0; k < array_size; k++) {
					cerr << host_arrays[step + k] << " ";
				}
				cerr << "]" << endl;
			}

			break;
		}
	}

	cout << "host_match: [";
	for (int i = 0; i < match_size; i++) {
		cout << host_match[i] << " ";
	}
	cout << "]" << endl;

	cout << "temp_match: [";
	for (int i = 0; i < match_size; i++) {
		cout << temp_match[i] << " ";
	}
	cout << "]" << endl;

	/***Free variables***/
	hipFree(device_arrays);
	hipFree(device_match);
	free(host_arrays);
	free(host_match);

	return 0;
}
