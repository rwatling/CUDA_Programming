/* This program searches "states" for matches in their arrays using CUDA
*
* Author: Robbie Watling
*/

#include "cuda_includes.h"
#include "array_match.h"
#include "shm_array_match.h"
#include <iostream>
#include <vector>
#include <utility>
#include <iostream>
#include <ctime>

using namespace std;

int main() {

	/***Variable Declarations***/
	int* host_arrays;
	int* device_arrays;
	int* host_match;
	int* device_match;

	int array_size;
	int match_size;
	int num_arrays;
	int NUM_THREADS;
	int NUM_BLOCKS;

	size_t one_t;
	size_t array_set_bytes;
	size_t match_bytes;

	hipError_t cuda_err;

	/***Initialization***/
	cin >> array_size;
	cin >> num_arrays;
	match_size = num_arrays;
	NUM_THREADS = num_arrays;
	NUM_BLOCKS = 1;

	// Host allocation
	one_t = (size_t) 1;
	array_set_bytes = (size_t) num_arrays * array_size * sizeof(int);
	match_bytes = (size_t) match_size * sizeof(int);

	host_arrays = (int*) calloc(one_t, array_set_bytes);
	host_match = (int*) calloc(one_t, match_bytes);


	if (host_arrays == NULL) {
		cerr << "Host arrays calloc failed\n" << endl;
		return -1;
	}

	if (host_match == NULL) {
		cerr << "Host match calloc failed\n" << endl;
		return -1;
	}

	//Device Allocation
	cuda_err = hipMalloc((void**)&device_arrays, array_set_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allocation for array set failed" << endl;
		return -1;
	}

	cuda_err = hipMalloc((void**)&device_match, match_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allcoation for match array failed" << endl;
		return -1;
	}

	/*** Search arrays and copy result back to host using shared memory***/
	shm_array_match <<<NUM_BLOCKS, NUM_THREADS >>> (device_arrays, device_match, num_arrays, array_size);

	//Copy match back to host
	hipMemcpy(host_match, device_match, match_bytes, hipMemcpyDeviceToHost);

	//Copy gpu arrays to host for verification
	hipMemcpy(host_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

	//Print arrays
	cout << "Original arrays:" << endl;
	for (int i = 0; i < num_arrays; i++) {
		int step = i * array_size;
		cout << "[";

		for (int j = 0; j < array_size; j++) {
			cout << host_arrays[step + j] << " " ;
		}

		cout << "]" << endl;
	}

	//Print match array
	cout << "Match array: [";
	for (int i = 0; i < match_size; i++) {
		cout << host_match[i] << " ";
	}
	cout << "]" << endl;

	//Zero out device memory
	hipMemset(device_arrays, 0, array_set_bytes);
	hipMemset(device_match, 0, match_bytes);

	/*** Search arrays and copy back to host using global memory ***/
	array_match <<<NUM_BLOCKS, NUM_THREADS >>> (device_arrays, device_match, num_arrays, array_size);

	//Copy match back to host
	hipMemcpy(host_match, device_match, match_bytes, hipMemcpyDeviceToHost);

	//Copy gpu arrays to host for verification
	hipMemcpy(host_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

	//Print arrays
	cout << "Original arrays:" << endl;
	for (int i = 0; i < num_arrays; i++) {
		int step = i * array_size;
		cout << "[";

		for (int j = 0; j < array_size; j++) {
			cout << host_arrays[step + j] << " " ;
		}

		cout << "]" << endl;
	}

	//Print match array
	cout << "Match array: [";
	for (int i = 0; i < match_size; i++) {
		cout << host_match[i] << " ";
	}
	cout << "]" << endl;

	/***Free variables***/
	hipFree(device_arrays);
	hipFree(device_match);
	free(host_arrays);
	free(host_match);

	return 0;
}
