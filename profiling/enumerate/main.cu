/* This program searches "states" for matches in their arrays using CUDA.
* This version for profiling memory type usages.
* Author: Robbie Watling
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <sys/time.h>

#define SHM_96_KB 98304
#define SHM_64_KB 65536
#define ARRAY_SIZE 8
#define WARP_SIZE 32

using namespace std;

// For shuffling host arrays
void shuffle(int *array, size_t n)
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  long int mytime = tp.tv_sec * 1000 + tp.tv_usec;
  srand(mytime);

  if (n > 1) {
      int i;
      for (i = 0; i < n - 1; i++){
        int j = i + rand() / (RAND_MAX / (n - i) + 1);
        int t = array[j];
        array[j] = array[i];
        array[i] = t;
      }
  }
}

__device__ void match(int* array2, int* next_arr1, int* next_arr2) {

  //TODO: Having an issue with no-matches
  for (int i = 0; i < ARRAY_SIZE; i++) {
    for (int j = 0; j < ARRAY_SIZE; j++) {
      if (array2[i] == next_arr1[j]) {
        array2[i] = next_arr2[j];
        break;
      }
    }
  }
}

__global__ void shm_array_match(int* global_arrays, int num_threads) {

	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	//int current_arr1[ARRAY_SIZE];
	//int current_arr2[ARRAY_SIZE];
	//int next_arr1[ARRAY_SIZE];
	//int next_arr2[ARRAY_SIZE];

  int size = ARRAY_SIZE; //STILL NEED TO UPDATE THIS FOR ENUMERATED VERSION

  //int arr1_index = 0;
	//int arr2_index = 0;

	//Retrieve global values from major operation
	//Assign the global values to registers for array_1 and array_2
	//Assign the initial global values to shared memory
	/*for (int i = 0; i < size; i++) {
		int arr1_index = (thread_id * 2 * size) + i;
		current_arr1[i] = global_arrays[arr1_index];

		int arr2_index = (thread_id * 2 * size) + size + i;
		current_arr2[i] = global_arrays[arr2_index];
	}*/

  //Enumerated version
  //Current array 1
  int current_a1_e0 = global_arrays[0];
  int current_a1_e1 = global_arrays[1];
  int current_a1_e2 = global_arrays[2];
  int current_a1_e3 = global_arrays[3];

  int current_a1_e4 = global_arrays[4];
  int current_a1_e5 = global_arrays[5];
  int current_a1_e6 = global_arrays[6];
  int current_a1_e7 = global_arrays[7];

  //Current array 2
  int current_a2_e0 = global_arrays[size + 0];
  int current_a2_e1 = global_arrays[size + 1];
  int current_a2_e2 = global_arrays[size + 2];
  int current_a2_e3 = global_arrays[size + 3];

  int current_a2_e4 = global_arrays[size + 4];
  int current_a2_e5 = global_arrays[size + 5];
  int current_a2_e6 = global_arrays[size + 6];
  int current_a2_e7 = global_arrays[size + 7];

  //Init next array 1
  int next_a1_e0 = 0;
  int next_a1_e1 = 0;
  int next_a1_e2 = 0;
  int next_a1_e3 = 0;

  int next_a1_e4 = 0;
  int next_a1_e5 = 0;
  int next_a1_e6 = 0;
  int next_a1_e7 = 0;

  //Init next array 2
  int next_a2_e0 = 0;
  int next_a2_e1 = 0;
  int next_a2_e2 = 0;
  int next_a2_e3 = 0;

  int next_a2_e4 = 0;
  int next_a2_e5 = 0;
  int next_a2_e6 = 0;
  int next_a2_e7 = 0;

	__syncthreads();

	// Tree like match reduction using shared memory
	for (int k = 1; k < num_threads; k = k << 1) {

		// If thread is a writer
		if ((thread_id % (k * 2)) == k) {

			//Write my first array to shared memory for communication
			/*for (int i = 0; i < size; i++) {
				arr1_index = (thread_id / (k * 2)) * 2 * size + i;
				shared_arrays[arr1_index] = current_arr1[i];
			}

			//Write my second array to shared memory for communication
			for (int i = 0; i < size; i++) {
				arr2_index = (thread_id / (k * 2)) * 2 * size + size + i;
				shared_arrays[arr2_index] = current_arr2[i];
			}*/

      //Write for current array 1
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 0] = current_a1_e0;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 1] = current_a1_e1;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 2] = current_a1_e2;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 3] = current_a1_e3;

      shared_arrays[(thread_id / (k * 2)) * 2 * size + 4] = current_a1_e4;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 5] = current_a1_e5;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 6] = current_a1_e6;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + 7] = current_a1_e7;

      //Write for current array 2
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 0] = current_a2_e0;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 1] = current_a2_e1;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 2] = current_a2_e2;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 3] = current_a2_e3;

      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 4] = current_a2_e4;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 5] = current_a2_e5;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 6] = current_a2_e6;
      shared_arrays[(thread_id / (k * 2)) * 2 * size + size + 7] = current_a2_e7;
		}

		__syncthreads();

		// If thread is a reader
		if ((thread_id % (k * 2) == 0)) {

			//Read my writers first array
			/*for (int i = 0; i < size; i++) {
				arr1_index = (thread_id / (k * 2)) * 2 * size + i;
				next_arr1[i] = shared_arrays[arr1_index];
			}

			//Read my writers second array
			for (int i = 0; i < size; i++) {
				arr2_index = (thread_id / (k * 2)) * 2 * size + size + i;
				next_arr2[i] = shared_arrays[arr2_index];
			}*/

      //Read next array 1
      next_a1_e0 = shared_arrays[(thread_id / (k*2)) * 2 * size + 0];
      next_a1_e1 = shared_arrays[(thread_id / (k*2)) * 2 * size + 1];
      next_a1_e2 = shared_arrays[(thread_id / (k*2)) * 2 * size + 2];
      next_a1_e3 = shared_arrays[(thread_id / (k*2)) * 2 * size + 3];

      next_a1_e4 = shared_arrays[(thread_id / (k*2)) * 2 * size + 4];
      next_a1_e5 = shared_arrays[(thread_id / (k*2)) * 2 * size + 5];
      next_a1_e6 = shared_arrays[(thread_id / (k*2)) * 2 * size + 6];
      next_a1_e7 = shared_arrays[(thread_id / (k*2)) * 2 * size + 7];

      //Read next array 2
      next_a2_e0 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 0];
      next_a2_e1 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 1];
      next_a2_e2 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 2];
      next_a2_e3 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 3];

      next_a2_e4 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 4];
      next_a2_e5 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 5];
      next_a2_e6 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 6];
      next_a2_e7 = shared_arrays[(thread_id / (k*2)) * 2 * size + size + 7];

			//match(current_arr2, next_arr1, next_arr2);

      //current_a2_e0
      if (current_a2_e0 == next_a1_e0) {
        current_a2_e0 = next_a2_e0;
      } else if (current_a2_e0 == next_a1_e1) {
        current_a2_e0 = next_a2_e1;
      } else if (current_a2_e0 == next_a1_e2) {
        current_a2_e0 = next_a2_e2;
      } else if (current_a2_e0 == next_a1_e3) {
        current_a2_e0 = next_a2_e3;
      } else if (current_a2_e0 == next_a1_e4) {
        current_a2_e0 = next_a2_e4;
      } else if (current_a2_e0 == next_a1_e5) {
        current_a2_e0 = next_a2_e5;
      } else if (current_a2_e0 == next_a1_e6) {
        current_a2_e0 = next_a2_e6;
      } else if (current_a2_e0 == next_a1_e7) {
        current_a2_e0 = next_a2_e7;
      }

      //current_a2_e1
      if (current_a2_e1 == next_a1_e0) {
        current_a2_e1 = next_a2_e0;
      } else if (current_a2_e1 == next_a1_e1) {
        current_a2_e1 = next_a2_e1;
      } else if (current_a2_e1 == next_a1_e2) {
        current_a2_e1 = next_a2_e2;
      } else if (current_a2_e1 == next_a1_e3) {
        current_a2_e1 = next_a2_e3;
      } else if (current_a2_e1 == next_a1_e4) {
        current_a2_e1 = next_a2_e4;
      } else if (current_a2_e1 == next_a1_e5) {
        current_a2_e1 = next_a2_e5;
      } else if (current_a2_e1 == next_a1_e6) {
        current_a2_e1 = next_a2_e6;
      } else if (current_a2_e1 == next_a1_e7) {
        current_a2_e1 = next_a2_e7;
      }

      //current_a2_e2
      if (current_a2_e2 == next_a1_e0) {
        current_a2_e2 = next_a2_e0;
      } else if (current_a2_e2 == next_a1_e1) {
        current_a2_e2 = next_a2_e1;
      } else if (current_a2_e2 == next_a1_e2) {
        current_a2_e2 = next_a2_e2;
      } else if (current_a2_e2 == next_a1_e3) {
        current_a2_e2 = next_a2_e3;
      } else if (current_a2_e2 == next_a1_e4) {
        current_a2_e2 = next_a2_e4;
      } else if (current_a2_e2 == next_a1_e5) {
        current_a2_e2 = next_a2_e5;
      } else if (current_a2_e2 == next_a1_e6) {
        current_a2_e2 = next_a2_e6;
      } else if (current_a2_e2 == next_a1_e7) {
        current_a2_e2 = next_a2_e7;
      }

      //current_a2_e3
      if (current_a2_e3 == next_a1_e0) {
        current_a2_e3 = next_a2_e0;
      } else if (current_a2_e3 == next_a1_e1) {
        current_a2_e3 = next_a2_e1;
      } else if (current_a2_e3 == next_a1_e2) {
        current_a2_e3 = next_a2_e2;
      } else if (current_a2_e3 == next_a1_e3) {
        current_a2_e3 = next_a2_e3;
      } else if (current_a2_e3 == next_a1_e4) {
        current_a2_e3 = next_a2_e4;
      } else if (current_a2_e3 == next_a1_e5) {
        current_a2_e3 = next_a2_e5;
      } else if (current_a2_e3 == next_a1_e6) {
        current_a2_e3 = next_a2_e6;
      } else if (current_a2_e3 == next_a1_e7) {
        current_a2_e3 = next_a2_e7;
      }

      //current_a2_e4
      /*if (current_a2_e4 == next_a1_e0) {
        current_a2_e4 = next_a2_e0;
      } else if (current_a2_e4 == next_a1_e1) {
        current_a2_e4 = next_a2_e1;
      } else if (current_a2_e4 == next_a1_e2) {
        current_a2_e4 = next_a2_e2;
      } else if (current_a2_e4 == next_a1_e3) {
        current_a2_e4 = next_a2_e3;
      } else if (current_a2_e4 == next_a1_e4) {
        current_a2_e4 = next_a2_e4;
      } else if (current_a2_e4 == next_a1_e5) {
        current_a2_e4 = next_a2_e5;
      } else if (current_a2_e4 == next_a1_e6) {
        current_a2_e4 = next_a2_e6;
      } else if (current_a2_e4 == next_a1_e7) {
        current_a2_e4 = next_a2_e7;
      }

      //current_a2_e5
      if (current_a2_e5 == next_a1_e0) {
        current_a2_e5 = next_a2_e0;
      } else if (current_a2_e5 == next_a1_e1) {
        current_a2_e5 = next_a2_e1;
      } else if (current_a2_e5 == next_a1_e2) {
        current_a2_e5 = next_a2_e2;
      } else if (current_a2_e5 == next_a1_e3) {
        current_a2_e5 = next_a2_e3;
      } else if (current_a2_e5 == next_a1_e4) {
        current_a2_e5 = next_a2_e4;
      } else if (current_a2_e5 == next_a1_e5) {
        current_a2_e5 = next_a2_e5;
      } else if (current_a2_e5 == next_a1_e6) {
        current_a2_e5 = next_a2_e6;
      } else if (current_a2_e5 == next_a1_e7) {
        current_a2_e5 = next_a2_e7;
      }

      //current_a2_e6
      if (current_a2_e6 == next_a1_e0) {
        current_a2_e6 = next_a2_e0;
      } else if (current_a2_e6 == next_a1_e1) {
        current_a2_e6 = next_a2_e1;
      } else if (current_a2_e6 == next_a1_e2) {
        current_a2_e6 = next_a2_e2;
      } else if (current_a2_e6 == next_a1_e3) {
        current_a2_e6 = next_a2_e3;
      } else if (current_a2_e6 == next_a1_e4) {
        current_a2_e6 = next_a2_e4;
      } else if (current_a2_e6 == next_a1_e5) {
        current_a2_e6 = next_a2_e5;
      } else if (current_a2_e6 == next_a1_e6) {
        current_a2_e6 = next_a2_e6;
      } else if (current_a2_e6 == next_a1_e7) {
        current_a2_e6 = next_a2_e7;
      }

      //current_a2_e7
      if (current_a2_e7 == next_a1_e0) {
        current_a2_e7 = next_a2_e0;
      } else if (current_a2_e7 == next_a1_e1) {
        current_a2_e7 = next_a2_e1;
      } else if (current_a2_e7 == next_a1_e2) {
        current_a2_e7 = next_a2_e2;
      } else if (current_a2_e7 == next_a1_e3) {
        current_a2_e7 = next_a2_e3;
      } else if (current_a2_e7 == next_a1_e4) {
        current_a2_e7 = next_a2_e4;
      } else if (current_a2_e7 == next_a1_e5) {
        current_a2_e7 = next_a2_e5;
      } else if (current_a2_e7 == next_a1_e6) {
        current_a2_e7 = next_a2_e6;
      } else if (current_a2_e7 == next_a1_e7) {
        current_a2_e7 = next_a2_e7;
      }*/
    }

    __syncthreads();
  }

	//Write shared memory to global memory for verification
	/*if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}*/

  if (thread_id == 0) {
    global_arrays[0] = current_a1_e0;
    global_arrays[1] = current_a1_e1;
    global_arrays[2] = current_a1_e2;
    global_arrays[3] = current_a1_e3;

    global_arrays[4] = current_a1_e4;
    global_arrays[5] = current_a1_e5;
    global_arrays[6] = current_a1_e6;
    global_arrays[7] = current_a1_e7;

    global_arrays[size + 0] = current_a2_e0;
    global_arrays[size + 1] = current_a2_e1;
    global_arrays[size + 2] = current_a2_e2;
    global_arrays[size + 3] = current_a2_e3;

    global_arrays[12] = current_a2_e4;
    global_arrays[13] = current_a2_e5;
    global_arrays[14] = current_a2_e6;
    global_arrays[15] = current_a2_e7;
  }
}

__global__ void shfl_array_match(int* global_arrays, int num_threads) {
  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	//int current_arr1[ARRAY_SIZE];
	//int current_arr2[ARRAY_SIZE];
  //int next_arr1[ARRAY_SIZE];
  //int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  //int arr1_index = 0;
  //int arr2_index = 0;
  unsigned int mask = 0xffffffff;

  //Stage 0: Retrieve from global memory
  /*for (int i = 0; i < size; i++) {
    arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }*/

  //Enumerated version
  //Current array 1
  int current_a1_e0 = global_arrays[0];
  int current_a1_e1 = global_arrays[1];
  int current_a1_e2 = global_arrays[2];
  int current_a1_e3 = global_arrays[3];

  int current_a1_e4 = global_arrays[4];
  int current_a1_e5 = global_arrays[5];
  int current_a1_e6 = global_arrays[6];
  int current_a1_e7 = global_arrays[7];

  //Current array 2
  int current_a2_e0 = global_arrays[size + 0];
  int current_a2_e1 = global_arrays[size + 1];
  int current_a2_e2 = global_arrays[size + 2];
  int current_a2_e3 = global_arrays[size + 3];

  int current_a2_e4 = global_arrays[size + 4];
  int current_a2_e5 = global_arrays[size + 5];
  int current_a2_e6 = global_arrays[size + 6];
  int current_a2_e7 = global_arrays[size + 7];

  //Init next array 1
  int next_a1_e0 = 0;
  int next_a1_e1 = 0;
  int next_a1_e2 = 0;
  int next_a1_e3 = 0;

  int next_a1_e4 = 0;
  int next_a1_e5 = 0;
  int next_a1_e6 = 0;
  int next_a1_e7 = 0;

  //Init next array 2
  int next_a2_e0 = 0;
  int next_a2_e1 = 0;
  int next_a2_e2 = 0;
  int next_a2_e3 = 0;

  int next_a2_e4 = 0;
  int next_a2_e5 = 0;
  int next_a2_e6 = 0;
  int next_a2_e7 = 0;

  //Stage 1: Match by shuffle arrays with tree like reduction
  for (int delta = 1; delta < WARP_SIZE; delta = delta << 1) {

    //Retrieve value from register from thread_id + delta
    /*for (int i = 0; i < size; i++) {
      next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
      next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
    }*/

    next_a1_e0 = __shfl_down_sync(mask, current_a1_e0, delta, WARP_SIZE);
    next_a2_e0 = __shfl_down_sync(mask, current_a2_e0, delta, WARP_SIZE);

    next_a1_e1 = __shfl_down_sync(mask, current_a1_e1, delta, WARP_SIZE);
    next_a2_e1 = __shfl_down_sync(mask, current_a2_e1, delta, WARP_SIZE);

    next_a1_e2 = __shfl_down_sync(mask, current_a1_e2, delta, WARP_SIZE);
    next_a2_e2 = __shfl_down_sync(mask, current_a2_e2, delta, WARP_SIZE);

    next_a1_e3 = __shfl_down_sync(mask, current_a1_e3, delta, WARP_SIZE);
    next_a2_e3 = __shfl_down_sync(mask, current_a2_e3, delta, WARP_SIZE);

    next_a1_e4 = __shfl_down_sync(mask, current_a1_e4, delta, WARP_SIZE);
    next_a2_e4 = __shfl_down_sync(mask, current_a2_e4, delta, WARP_SIZE);

    next_a1_e5 = __shfl_down_sync(mask, current_a1_e5, delta, WARP_SIZE);
    next_a2_e5 = __shfl_down_sync(mask, current_a2_e5, delta, WARP_SIZE);

    next_a1_e6 = __shfl_down_sync(mask, current_a1_e6, delta, WARP_SIZE);
    next_a2_e6 = __shfl_down_sync(mask, current_a2_e6, delta, WARP_SIZE);

    next_a1_e7 = __shfl_down_sync(mask, current_a1_e7, delta, WARP_SIZE);
    next_a2_e7 = __shfl_down_sync(mask, current_a2_e7, delta, WARP_SIZE);

    if ((thread_id % (delta * 2)) == 0) {
      //current_a2_e0
      if (current_a2_e0 == next_a1_e0) {
        current_a2_e0 = next_a2_e0;
      } else if (current_a2_e0 == next_a1_e1) {
        current_a2_e0 = next_a2_e1;
      } else if (current_a2_e0 == next_a1_e2) {
        current_a2_e0 = next_a2_e2;
      } else if (current_a2_e0 == next_a1_e3) {
        current_a2_e0 = next_a2_e3;
      } else if (current_a2_e0 == next_a1_e4) {
        current_a2_e0 = next_a2_e4;
      } else if (current_a2_e0 == next_a1_e5) {
        current_a2_e0 = next_a2_e5;
      } else if (current_a2_e0 == next_a1_e6) {
        current_a2_e0 = next_a2_e6;
      } else if (current_a2_e0 == next_a1_e7) {
        current_a2_e0 = next_a2_e7;
      }

      //current_a2_e1
      if (current_a2_e1 == next_a1_e0) {
        current_a2_e1 = next_a2_e0;
      } else if (current_a2_e1 == next_a1_e1) {
        current_a2_e1 = next_a2_e1;
      } else if (current_a2_e1 == next_a1_e2) {
        current_a2_e1 = next_a2_e2;
      } else if (current_a2_e1 == next_a1_e3) {
        current_a2_e1 = next_a2_e3;
      } else if (current_a2_e1 == next_a1_e4) {
        current_a2_e1 = next_a2_e4;
      } else if (current_a2_e1 == next_a1_e5) {
        current_a2_e1 = next_a2_e5;
      } else if (current_a2_e1 == next_a1_e6) {
        current_a2_e1 = next_a2_e6;
      } else if (current_a2_e1 == next_a1_e7) {
        current_a2_e1 = next_a2_e7;
      }

      //current_a2_e2
      if (current_a2_e2 == next_a1_e0) {
        current_a2_e2 = next_a2_e0;
      } else if (current_a2_e2 == next_a1_e1) {
        current_a2_e2 = next_a2_e1;
      } else if (current_a2_e2 == next_a1_e2) {
        current_a2_e2 = next_a2_e2;
      } else if (current_a2_e2 == next_a1_e3) {
        current_a2_e2 = next_a2_e3;
      } else if (current_a2_e2 == next_a1_e4) {
        current_a2_e2 = next_a2_e4;
      } else if (current_a2_e2 == next_a1_e5) {
        current_a2_e2 = next_a2_e5;
      } else if (current_a2_e2 == next_a1_e6) {
        current_a2_e2 = next_a2_e6;
      } else if (current_a2_e2 == next_a1_e7) {
        current_a2_e2 = next_a2_e7;
      }

      //current_a2_e3
      if (current_a2_e3 == next_a1_e0) {
        current_a2_e3 = next_a2_e0;
      } else if (current_a2_e3 == next_a1_e1) {
        current_a2_e3 = next_a2_e1;
      } else if (current_a2_e3 == next_a1_e2) {
        current_a2_e3 = next_a2_e2;
      } else if (current_a2_e3 == next_a1_e3) {
        current_a2_e3 = next_a2_e3;
      } else if (current_a2_e3 == next_a1_e4) {
        current_a2_e3 = next_a2_e4;
      } else if (current_a2_e3 == next_a1_e5) {
        current_a2_e3 = next_a2_e5;
      } else if (current_a2_e3 == next_a1_e6) {
        current_a2_e3 = next_a2_e6;
      } else if (current_a2_e3 == next_a1_e7) {
        current_a2_e3 = next_a2_e7;
      }

      //current_a2_e4
      if (current_a2_e4 == next_a1_e0) {
        current_a2_e4 = next_a2_e0;
      } else if (current_a2_e4 == next_a1_e1) {
        current_a2_e4 = next_a2_e1;
      } else if (current_a2_e4 == next_a1_e2) {
        current_a2_e4 = next_a2_e2;
      } else if (current_a2_e4 == next_a1_e3) {
        current_a2_e4 = next_a2_e3;
      } else if (current_a2_e4 == next_a1_e4) {
        current_a2_e4 = next_a2_e4;
      } else if (current_a2_e4 == next_a1_e5) {
        current_a2_e4 = next_a2_e5;
      } else if (current_a2_e4 == next_a1_e6) {
        current_a2_e4 = next_a2_e6;
      } else if (current_a2_e4 == next_a1_e7) {
        current_a2_e4 = next_a2_e7;
      }

      //current_a2_e5
      if (current_a2_e5 == next_a1_e0) {
        current_a2_e5 = next_a2_e0;
      } else if (current_a2_e5 == next_a1_e1) {
        current_a2_e5 = next_a2_e1;
      } else if (current_a2_e5 == next_a1_e2) {
        current_a2_e5 = next_a2_e2;
      } else if (current_a2_e5 == next_a1_e3) {
        current_a2_e5 = next_a2_e3;
      } else if (current_a2_e5 == next_a1_e4) {
        current_a2_e5 = next_a2_e4;
      } else if (current_a2_e5 == next_a1_e5) {
        current_a2_e5 = next_a2_e5;
      } else if (current_a2_e5 == next_a1_e6) {
        current_a2_e5 = next_a2_e6;
      } else if (current_a2_e5 == next_a1_e7) {
        current_a2_e5 = next_a2_e7;
      }

      //current_a2_e6
      if (current_a2_e6 == next_a1_e0) {
        current_a2_e6 = next_a2_e0;
      } else if (current_a2_e6 == next_a1_e1) {
        current_a2_e6 = next_a2_e1;
      } else if (current_a2_e6 == next_a1_e2) {
        current_a2_e6 = next_a2_e2;
      } else if (current_a2_e6 == next_a1_e3) {
        current_a2_e6 = next_a2_e3;
      } else if (current_a2_e6 == next_a1_e4) {
        current_a2_e6 = next_a2_e4;
      } else if (current_a2_e6 == next_a1_e5) {
        current_a2_e6 = next_a2_e5;
      } else if (current_a2_e6 == next_a1_e6) {
        current_a2_e6 = next_a2_e6;
      } else if (current_a2_e6 == next_a1_e7) {
        current_a2_e6 = next_a2_e7;
      }

      //current_a2_e7
      if (current_a2_e7 == next_a1_e0) {
        current_a2_e7 = next_a2_e0;
      } else if (current_a2_e7 == next_a1_e1) {
        current_a2_e7 = next_a2_e1;
      } else if (current_a2_e7 == next_a1_e2) {
        current_a2_e7 = next_a2_e2;
      } else if (current_a2_e7 == next_a1_e3) {
        current_a2_e7 = next_a2_e3;
      } else if (current_a2_e7 == next_a1_e4) {
        current_a2_e7 = next_a2_e4;
      } else if (current_a2_e7 == next_a1_e5) {
        current_a2_e7 = next_a2_e5;
      } else if (current_a2_e7 == next_a1_e6) {
        current_a2_e7 = next_a2_e6;
      } else if (current_a2_e7 == next_a1_e7) {
        current_a2_e7 = next_a2_e7;
      }
    }
  }

  if (num_threads > WARP_SIZE) {

    //Stage 2: Warp thread 0 write warp shuffle result to shared memory
    if ((thread_id % WARP_SIZE) == 0) {

      /*for(int i = 0; i < size; i++) {
        arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
        shared_arrays[arr1_index] = current_arr1[i];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = ((thread_id / WARP_SIZE) * 2 * size) + size + i;
        shared_arrays[arr2_index] = current_arr2[i];
      }*/

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 0] = current_a1_e0;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 0] = current_a2_e0;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 1] = current_a1_e1;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 1] = current_a2_e1;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 2] = current_a1_e2;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 2] = current_a2_e2;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 3] = current_a1_e3;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 3] = current_a2_e3;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 4] = current_a1_e4;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 4] = current_a2_e4;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 5] = current_a1_e5;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 5] = current_a2_e5;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 6] = current_a1_e6;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 6] = current_a2_e6;

      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + 7] = current_a1_e7;
      shared_arrays[((thread_id / WARP_SIZE) * 2 * size) + size + 7] = current_a2_e7;
    }

    __syncthreads();

    //Stage 3: Read all warps 0 thread from shared memory
    if (thread_id < WARP_SIZE) {
      current_a1_e0 = shared_arrays[(thread_id * 2 * size) + 0];
      current_a2_e0 = shared_arrays[(thread_id * 2 * size) + size + 0];

      current_a1_e1 = shared_arrays[(thread_id * 2 * size) + 1];
      current_a2_e1 = shared_arrays[(thread_id * 2 * size) + size + 1];

      current_a1_e2 = shared_arrays[(thread_id * 2 * size) + 2];
      current_a2_e2 = shared_arrays[(thread_id * 2 * size) + size + 2];

      current_a1_e3 = shared_arrays[(thread_id * 2 * size) + 3];
      current_a2_e3 = shared_arrays[(thread_id * 2 * size) + size + 3];

      current_a1_e4 = shared_arrays[(thread_id * 2 * size) + 4];
      current_a2_e4 = shared_arrays[(thread_id * 2 * size) + size + 4];

      current_a1_e5 = shared_arrays[(thread_id * 2 * size) + 5];
      current_a2_e5 = shared_arrays[(thread_id * 2 * size) + size + 5];

      current_a1_e6 = shared_arrays[(thread_id * 2 * size) + 6];
      current_a2_e6 = shared_arrays[(thread_id * 2 * size) + size + 6];

      current_a1_e7 = shared_arrays[(thread_id * 2 * size) + 7];
      current_a2_e7 = shared_arrays[(thread_id * 2 * size) + size + 7];
    }

    __syncthreads();

    //Stage 4: Shuffle again
    if (thread_id < WARP_SIZE) {

      // Tree like reduction, notice for loop condition
      for (int delta = 1; delta < (num_threads / WARP_SIZE); delta = delta << 1) {

        next_a1_e0 = __shfl_down_sync(mask, current_a1_e0, delta, WARP_SIZE);
        next_a2_e0 = __shfl_down_sync(mask, current_a2_e0, delta, WARP_SIZE);

        next_a1_e1 = __shfl_down_sync(mask, current_a1_e1, delta, WARP_SIZE);
        next_a2_e1 = __shfl_down_sync(mask, current_a2_e1, delta, WARP_SIZE);

        next_a1_e2 = __shfl_down_sync(mask, current_a1_e2, delta, WARP_SIZE);
        next_a2_e2 = __shfl_down_sync(mask, current_a2_e2, delta, WARP_SIZE);

        next_a1_e3 = __shfl_down_sync(mask, current_a1_e3, delta, WARP_SIZE);
        next_a2_e3 = __shfl_down_sync(mask, current_a2_e3, delta, WARP_SIZE);

        next_a1_e4 = __shfl_down_sync(mask, current_a1_e4, delta, WARP_SIZE);
        next_a2_e4 = __shfl_down_sync(mask, current_a2_e4, delta, WARP_SIZE);

        next_a1_e5 = __shfl_down_sync(mask, current_a1_e5, delta, WARP_SIZE);
        next_a2_e5 = __shfl_down_sync(mask, current_a2_e5, delta, WARP_SIZE);

        next_a1_e6 = __shfl_down_sync(mask, current_a1_e6, delta, WARP_SIZE);
        next_a2_e6 = __shfl_down_sync(mask, current_a2_e6, delta, WARP_SIZE);

        next_a1_e7 = __shfl_down_sync(mask, current_a1_e7, delta, WARP_SIZE);
        next_a2_e7 = __shfl_down_sync(mask, current_a2_e7, delta, WARP_SIZE);

        if ((thread_id % (delta * 2)) == 0) {
          //current_a2_e0
          if (current_a2_e0 == next_a1_e0) {
            current_a2_e0 = next_a2_e0;
          } else if (current_a2_e0 == next_a1_e1) {
            current_a2_e0 = next_a2_e1;
          } else if (current_a2_e0 == next_a1_e2) {
            current_a2_e0 = next_a2_e2;
          } else if (current_a2_e0 == next_a1_e3) {
            current_a2_e0 = next_a2_e3;
          } else if (current_a2_e0 == next_a1_e4) {
            current_a2_e0 = next_a2_e4;
          } else if (current_a2_e0 == next_a1_e5) {
            current_a2_e0 = next_a2_e5;
          } else if (current_a2_e0 == next_a1_e6) {
            current_a2_e0 = next_a2_e6;
          } else if (current_a2_e0 == next_a1_e7) {
            current_a2_e0 = next_a2_e7;
          }

          //current_a2_e1
          if (current_a2_e1 == next_a1_e0) {
            current_a2_e1 = next_a2_e0;
          } else if (current_a2_e1 == next_a1_e1) {
            current_a2_e1 = next_a2_e1;
          } else if (current_a2_e1 == next_a1_e2) {
            current_a2_e1 = next_a2_e2;
          } else if (current_a2_e1 == next_a1_e3) {
            current_a2_e1 = next_a2_e3;
          } else if (current_a2_e1 == next_a1_e4) {
            current_a2_e1 = next_a2_e4;
          } else if (current_a2_e1 == next_a1_e5) {
            current_a2_e1 = next_a2_e5;
          } else if (current_a2_e1 == next_a1_e6) {
            current_a2_e1 = next_a2_e6;
          } else if (current_a2_e1 == next_a1_e7) {
            current_a2_e1 = next_a2_e7;
          }

          //current_a2_e2
          if (current_a2_e2 == next_a1_e0) {
            current_a2_e2 = next_a2_e0;
          } else if (current_a2_e2 == next_a1_e1) {
            current_a2_e2 = next_a2_e1;
          } else if (current_a2_e2 == next_a1_e2) {
            current_a2_e2 = next_a2_e2;
          } else if (current_a2_e2 == next_a1_e3) {
            current_a2_e2 = next_a2_e3;
          } else if (current_a2_e2 == next_a1_e4) {
            current_a2_e2 = next_a2_e4;
          } else if (current_a2_e2 == next_a1_e5) {
            current_a2_e2 = next_a2_e5;
          } else if (current_a2_e2 == next_a1_e6) {
            current_a2_e2 = next_a2_e6;
          } else if (current_a2_e2 == next_a1_e7) {
            current_a2_e2 = next_a2_e7;
          }

          //current_a2_e3
          if (current_a2_e3 == next_a1_e0) {
            current_a2_e3 = next_a2_e0;
          } else if (current_a2_e3 == next_a1_e1) {
            current_a2_e3 = next_a2_e1;
          } else if (current_a2_e3 == next_a1_e2) {
            current_a2_e3 = next_a2_e2;
          } else if (current_a2_e3 == next_a1_e3) {
            current_a2_e3 = next_a2_e3;
          } else if (current_a2_e3 == next_a1_e4) {
            current_a2_e3 = next_a2_e4;
          } else if (current_a2_e3 == next_a1_e5) {
            current_a2_e3 = next_a2_e5;
          } else if (current_a2_e3 == next_a1_e6) {
            current_a2_e3 = next_a2_e6;
          } else if (current_a2_e3 == next_a1_e7) {
            current_a2_e3 = next_a2_e7;
          }

          //current_a2_e4
          if (current_a2_e4 == next_a1_e0) {
            current_a2_e4 = next_a2_e0;
          } else if (current_a2_e4 == next_a1_e1) {
            current_a2_e4 = next_a2_e1;
          } else if (current_a2_e4 == next_a1_e2) {
            current_a2_e4 = next_a2_e2;
          } else if (current_a2_e4 == next_a1_e3) {
            current_a2_e4 = next_a2_e3;
          } else if (current_a2_e4 == next_a1_e4) {
            current_a2_e4 = next_a2_e4;
          } else if (current_a2_e4 == next_a1_e5) {
            current_a2_e4 = next_a2_e5;
          } else if (current_a2_e4 == next_a1_e6) {
            current_a2_e4 = next_a2_e6;
          } else if (current_a2_e4 == next_a1_e7) {
            current_a2_e4 = next_a2_e7;
          }

          //current_a2_e5
          if (current_a2_e5 == next_a1_e0) {
            current_a2_e5 = next_a2_e0;
          } else if (current_a2_e5 == next_a1_e1) {
            current_a2_e5 = next_a2_e1;
          } else if (current_a2_e5 == next_a1_e2) {
            current_a2_e5 = next_a2_e2;
          } else if (current_a2_e5 == next_a1_e3) {
            current_a2_e5 = next_a2_e3;
          } else if (current_a2_e5 == next_a1_e4) {
            current_a2_e5 = next_a2_e4;
          } else if (current_a2_e5 == next_a1_e5) {
            current_a2_e5 = next_a2_e5;
          } else if (current_a2_e5 == next_a1_e6) {
            current_a2_e5 = next_a2_e6;
          } else if (current_a2_e5 == next_a1_e7) {
            current_a2_e5 = next_a2_e7;
          }

          //current_a2_e6
          if (current_a2_e6 == next_a1_e0) {
            current_a2_e6 = next_a2_e0;
          } else if (current_a2_e6 == next_a1_e1) {
            current_a2_e6 = next_a2_e1;
          } else if (current_a2_e6 == next_a1_e2) {
            current_a2_e6 = next_a2_e2;
          } else if (current_a2_e6 == next_a1_e3) {
            current_a2_e6 = next_a2_e3;
          } else if (current_a2_e6 == next_a1_e4) {
            current_a2_e6 = next_a2_e4;
          } else if (current_a2_e6 == next_a1_e5) {
            current_a2_e6 = next_a2_e5;
          } else if (current_a2_e6 == next_a1_e6) {
            current_a2_e6 = next_a2_e6;
          } else if (current_a2_e6 == next_a1_e7) {
            current_a2_e6 = next_a2_e7;
          }

          //current_a2_e7
          if (current_a2_e7 == next_a1_e0) {
            current_a2_e7 = next_a2_e0;
          } else if (current_a2_e7 == next_a1_e1) {
            current_a2_e7 = next_a2_e1;
          } else if (current_a2_e7 == next_a1_e2) {
            current_a2_e7 = next_a2_e2;
          } else if (current_a2_e7 == next_a1_e3) {
            current_a2_e7 = next_a2_e3;
          } else if (current_a2_e7 == next_a1_e4) {
            current_a2_e7 = next_a2_e4;
          } else if (current_a2_e7 == next_a1_e5) {
            current_a2_e7 = next_a2_e5;
          } else if (current_a2_e7 == next_a1_e6) {
            current_a2_e7 = next_a2_e6;
          } else if (current_a2_e7 == next_a1_e7) {
            current_a2_e7 = next_a2_e7;
          }
      }
    }
  }
}

  //Stage 5: Write back to global memory
  if (thread_id == 0) {
		/*for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}*/

    global_arrays[0] = current_a1_e0;
    global_arrays[1] = current_a1_e1;
    global_arrays[2] = current_a1_e2;
    global_arrays[3] = current_a1_e3;

    global_arrays[4] = current_a1_e4;
    global_arrays[5] = current_a1_e5;
    global_arrays[6] = current_a1_e6;
    global_arrays[7] = current_a1_e7;

    global_arrays[size + 0] = current_a2_e0;
    global_arrays[size + 1] = current_a2_e1;
    global_arrays[size + 2] = current_a2_e2;
    global_arrays[size + 3] = current_a2_e3;

    global_arrays[12] = current_a2_e4;
    global_arrays[13] = current_a2_e5;
    global_arrays[14] = current_a2_e6;
    global_arrays[15] = current_a2_e7;
	}
}

void cpu_array_match(int* arrays, int num_threads, int array_size) {
  for (int i = 1; i < num_threads; i++) {
    int* next_arr1 = arrays + (i * 2 * array_size);
    int* next_arr2 = arrays + (i * 2 * array_size) + array_size;

    for (int j = 0; j < array_size; j++) {

      for (int k = 0; k < array_size; k++) {
        if (arrays[array_size + j] == next_arr1[k]) {
          arrays[array_size + j] = next_arr2[k];
          break;
        }
      }

    }
  }

}

int main(int argc, char** argv) {

	/***Variable Declarations***/
  float milliseconds;

  int* host_arrays;
  int* experiment1_arrays;
	int* device_arrays;

	int array_size;
  int num_arrays;
	int num_threads;
	int num_blocks;
  int share_size;
  int debug;

	size_t one_t;
	size_t array_set_bytes;

  hipEvent_t start, stop;
  hipError_t cuda_err;

	/*** Read args ***/
	if (argc < 3) {
		cerr << "./gpu_match num_operating_threads debug(1 or 0)" << endl;
		return -1;
	}

	/***Initialization***/
	array_size = ARRAY_SIZE;
	num_arrays = atoi(argv[1]);
  debug = (atoi(argv[2]));
  num_threads = num_arrays;
	num_blocks = 1;
  share_size = SHM_64_KB;


	//Host allocation
	one_t = (size_t) 1;
	array_set_bytes = (size_t) num_threads * array_size * 2 * sizeof(int);
  host_arrays = (int*) calloc(one_t, array_set_bytes);

	if (host_arrays == NULL) {
		cerr << "Host arrays calloc failed\n" << endl;
		return -1;
	}

  //Experiment arrays allocation
  experiment1_arrays = (int*) calloc(one_t, array_set_bytes);

  if (experiment1_arrays == NULL) {
		cerr << "experiment1 arrays calloc failed\n" << endl;
		return -1;
	}

	//Device Allocation
	cuda_err = hipMalloc((void**)&device_arrays, array_set_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allocation for array set failed" << endl;
		return -1;
	}

  //Fill in host arrays to emulate major operation
  for(int i = 0; i < num_threads; i++) {

    //Start array
		for(int j = 0; j < array_size; j++) {
      if (i != 0) {
        host_arrays[(i * array_size * 2) + j] = j;
      }
		}

    if (i != 0) { shuffle(host_arrays + (i * array_size * 2), array_size); }

    //End array
    for(int j = array_size; j < array_size * 2; j++) {
      host_arrays[(i * array_size * 2) + j] = j % array_size;
		}

    shuffle(host_arrays + (i * array_size * 2) + array_size, array_size);
	}

  //Print arrays before matching
  if (debug) {
    for(int i = 0; i < num_threads; i++) {

      cout << "Arrays " << i << ": [";

  		for(int j = 0; j < array_size * 2; j++) {
  			cout << host_arrays[(i * array_size * 2) + j] << " ";

        if (j == array_size - 1) { cout << "]\t["; }
  		}

      cout << "]" << endl;
  	}
  }

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  share_size = SHM_96_KB;
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {

    if (debug) { cerr << endl << "Dynamic shared memory size of 96kb for array set failed, trying 64kb" << endl; }
    share_size = SHM_64_KB;

    cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

    if (cuda_err != hipSuccess) {

      if (debug) { cerr << "Dynamic shared memory size of 64000 for array set failed. Exiting program..." << endl; }

      return -1;
    }
  }

  if (debug) {
    cout << endl << "***Experiment1***" << endl;
    cout << "--------------------KERNEL CALL--------------------" << endl;
  }

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Kernel call
  shm_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy device arrays back to host
  hipMemcpy(experiment1_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  cout << 1 << "," << num_threads << "," << array_size << "," << milliseconds << endl;


  /************************Experiment 1***************************************/

  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  share_size = SHM_96_KB;
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {

    if (debug) { cerr << endl << "Dynamic shared memory size of 96kb for array set failed, trying 64kb" << endl; }
    share_size = SHM_64_KB;

    cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

    if (cuda_err != hipSuccess) {

      if (debug) { cerr << "Dynamic shared memory size of 64000 for array set failed. Exiting program..." << endl; }

      return -1;
    }
	}

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  if (debug) {
    cout << endl << "***Experiment2***" << endl;
    cout << "--------------------KERNEL CALL--------------------" << endl;
  }

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Kernel call
  shfl_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy device arrays back to host
  hipMemcpy(experiment1_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  cout << 1 << "," << num_threads << "," << array_size << "," << milliseconds << endl;

	/***Free variables***/
	hipFree(device_arrays);
	free(host_arrays);
  free(experiment1_arrays);

	return 0;
}
