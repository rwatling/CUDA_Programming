/* This program searches "states" for matches in their arrays using CUDA.
* This version for profiling memory type usages.
* Author: Robbie Watling
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <sys/time.h>

#define SHM_96_KB 98304
#define SHM_64_KB 65536
#define ARRAY_SIZE 23
#define WARP_SIZE 32

using namespace std;

// For shuffling host arrays
void shuffle(int *array, size_t n)
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  long int mytime = tp.tv_sec * 1000 + tp.tv_usec;
  srand(mytime);

  if (n > 1) {
      int i;
      for (i = 0; i < n - 1; i++){
        int j = i + rand() / (RAND_MAX / (n - i) + 1);
        int t = array[j];
        array[j] = array[i];
        array[i] = t;
      }
  }
}

__device__ void match(int* array2, int* next_arr1, int* next_arr2) {

  //TODO: Having an issue with no-matches
  for (int i = 0; i < ARRAY_SIZE; i++) {
    for (int j = 0; j < ARRAY_SIZE; j++) {
      if (array2[i] == next_arr1[j]) {
        array2[i] = next_arr2[j];
        break;
      }
    }
  }
}

__global__ void shm_array_match(int* global_arrays, int num_threads) {

	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
	int next_arr1[ARRAY_SIZE];
	int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
	int arr1_index = 0;
	int arr2_index = 0;

	//Retrieve global values from major operation
	//Assign the global values to registers for array_1 and array_2
	//Assign the initial global values to shared memory
	for (int i = 0; i < size; i++) {
		int arr1_index = (thread_id * 2 * size) + i;
		current_arr1[i] = global_arrays[arr1_index];

		int arr2_index = (thread_id * 2 * size) + size + i;
		current_arr2[i] = global_arrays[arr2_index];
	}

	__syncthreads();

	// Tree like match reduction using shared memory
	for (int k = 1; k < num_threads; k = k << 1) {

		// If thread is a writer
		if ((thread_id % (k * 2)) == k) {

			//Write my first array to shared memory for communication
			for (int i = 0; i < size; i++) {
				arr1_index = (thread_id / (k * 2)) * 2 * size + i;
				shared_arrays[arr1_index] = current_arr1[i];
			}

			//Write my second array to shared memory for communication
			for (int i = 0; i < size; i++) {
				arr2_index = (thread_id / (k * 2)) * 2 * size + size + i;
				shared_arrays[arr2_index] = current_arr2[i];
			}
		}

		__syncthreads();

		// If thread is a reader
		if ((thread_id % (k * 2) == 0)) {

			//Read my writers first array
			for (int i = 0; i < size; i++) {
				arr1_index = (thread_id / (k * 2)) * 2 * size + i;
				next_arr1[i] = shared_arrays[arr1_index];
			}

			//Read my writers second array
			for (int i = 0; i < size; i++) {
				arr2_index = (thread_id / (k * 2)) * 2 * size + size + i;
				next_arr2[i] = shared_arrays[arr2_index];
			}

			match(current_arr2, next_arr1, next_arr2);
		}

		__syncthreads();
	}

	//Write shared memory to global memory for verification
	if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}

}

__global__ void shfl_array_match(int* global_arrays, int num_threads) {
  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
  int next_arr1[ARRAY_SIZE];
  int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  int arr1_index = 0;
  int arr2_index = 0;
  unsigned int mask = 0xffffffff;

  //Stage 0: Retrieve from global memory
  for (int i = 0; i < size; i++) {
    arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }

  //Stage 1: Match by shuffle arrays with tree like reduction
  for (int delta = 1; delta < WARP_SIZE; delta = delta << 1) {

    //Retrieve value from register from thread_id + delta
    for (int i = 0; i < size; i++) {
      next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
      next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
    }

    if ((thread_id % (delta * 2)) == 0) {
      match(current_arr2, next_arr1, next_arr2);
    }
  }

  if (num_threads > WARP_SIZE) {

    //Stage 2: Warp thread 0 write warp shuffle result to shared memory
    if ((thread_id % WARP_SIZE) == 0) {

      for(int i = 0; i < size; i++) {
        arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
        shared_arrays[arr1_index] = current_arr1[i];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = ((thread_id / WARP_SIZE) * 2 * size) + size + i;
        shared_arrays[arr2_index] = current_arr2[i];
      }
    }

    __syncthreads();

    //Stage 3: Read all warps 0 thread from shared memory
    if (thread_id < WARP_SIZE) {
      for(int i = 0; i < size; i++) {
        arr1_index = (thread_id * 2 * size) + i;
        current_arr1[i] = shared_arrays[arr1_index];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = (thread_id * 2 * size) + size + i;
        current_arr2[i] = shared_arrays[arr2_index];
      }
    }

    __syncthreads();

    //Stage 4: Shuffle again
    if (thread_id < WARP_SIZE) {

      // Tree like reduction, notice for loop condition
      for (int delta = 1; delta < (num_threads / WARP_SIZE); delta = delta << 1) {

        //Retrieve value from register from thread_id + delta
        for (int i = 0; i < size; i++) {
          next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
          next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
        }

        if ((thread_id % (delta * 2)) == 0) {
          match(current_arr2, next_arr1, next_arr2);
        }
      }
    }
  }

  //Stage 5: Write back to global memory
  if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}
}

void cpu_array_match(int* arrays, int num_threads, int array_size) {
  for (int i = 1; i < num_threads; i++) {
    int* next_arr1 = arrays + (i * 2 * array_size);
    int* next_arr2 = arrays + (i * 2 * array_size) + array_size;

    for (int j = 0; j < array_size; j++) {

      for (int k = 0; k < array_size; k++) {
        if (arrays[array_size + j] == next_arr1[k]) {
          arrays[array_size + j] = next_arr2[k];
          break;
        }
      }

    }
  }

}

int main(int argc, char** argv) {

	/***Variable Declarations***/
  float milliseconds;

  int* host_arrays;
  int* experiment1_arrays;
	int* device_arrays;

	int array_size;
  int num_arrays;
	int num_threads;
	int num_blocks;
  int share_size;
  int debug;

	size_t one_t;
	size_t array_set_bytes;

  hipEvent_t start, stop;
  hipError_t cuda_err;

	/*** Read args ***/
	if (argc < 3) {
		cerr << "./gpu_match num_operating_threads debug(1 or 0)" << endl;
		return -1;
	}

	/***Initialization***/
	array_size = ARRAY_SIZE;
	num_arrays = atoi(argv[1]);
  debug = (atoi(argv[2]));
  num_threads = num_arrays;
	num_blocks = 1;
  share_size = SHM_64_KB;


	//Host allocation
	one_t = (size_t) 1;
	array_set_bytes = (size_t) num_threads * array_size * 2 * sizeof(int);
  host_arrays = (int*) calloc(one_t, array_set_bytes);

	if (host_arrays == NULL) {
		cerr << "Host arrays calloc failed\n" << endl;
		return -1;
	}

  //Experiment arrays allocation
  experiment1_arrays = (int*) calloc(one_t, array_set_bytes);

  if (experiment1_arrays == NULL) {
		cerr << "experiment1 arrays calloc failed\n" << endl;
		return -1;
	}

	//Device Allocation
	cuda_err = hipMalloc((void**)&device_arrays, array_set_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allocation for array set failed" << endl;
		return -1;
	}

  //Fill in host arrays to emulate major operation
  for(int i = 0; i < num_threads; i++) {

    //Start array
		for(int j = 0; j < array_size; j++) {
      if (i != 0) {
        host_arrays[(i * array_size * 2) + j] = j;
      }
		}

    if (i != 0) { shuffle(host_arrays + (i * array_size * 2), array_size); }

    //End array
    for(int j = array_size; j < array_size * 2; j++) {
      host_arrays[(i * array_size * 2) + j] = j % array_size;
		}

    shuffle(host_arrays + (i * array_size * 2) + array_size, array_size);
	}

  //Print arrays before matching
  if (debug) {
    for(int i = 0; i < num_threads; i++) {

      cout << "Arrays " << i << ": [";

  		for(int j = 0; j < array_size * 2; j++) {
  			cout << host_arrays[(i * array_size * 2) + j] << " ";

        if (j == array_size - 1) { cout << "]\t["; }
  		}

      cout << "]" << endl;
  	}
  }

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  share_size = SHM_96_KB;
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {

    if (debug) { cerr << endl << "Dynamic shared memory size of 96kb for array set failed, trying 64kb" << endl; }
    share_size = SHM_64_KB;

    cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

    if (cuda_err != hipSuccess) {

      if (debug) { cerr << "Dynamic shared memory size of 64000 for array set failed. Exiting program..." << endl; }

      return -1;
    }
  }

  if (debug) {
    cout << endl << "***Experiment1***" << endl;
    cout << "--------------------KERNEL CALL--------------------" << endl;
  }

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Kernel call
  shm_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy device arrays back to host
  hipMemcpy(experiment1_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  cout << 1 << "," << num_threads << "," << array_size << "," << milliseconds << endl;


  /************************Experiment 1***************************************/

  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  share_size = SHM_96_KB;
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {

    if (debug) { cerr << endl << "Dynamic shared memory size of 96kb for array set failed, trying 64kb" << endl; }
    share_size = SHM_64_KB;

    cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

    if (cuda_err != hipSuccess) {

      if (debug) { cerr << "Dynamic shared memory size of 64000 for array set failed. Exiting program..." << endl; }

      return -1;
    }
	}

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  if (debug) {
    cout << endl << "***Experiment2***" << endl;
    cout << "--------------------KERNEL CALL--------------------" << endl;
  }

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Kernel call
  shfl_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy device arrays back to host
  hipMemcpy(experiment1_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  cout << 1 << "," << num_threads << "," << array_size << "," << milliseconds << endl;

	/***Free variables***/
	hipFree(device_arrays);
	free(host_arrays);
  free(experiment1_arrays);

	return 0;
}
