#include "hip/hip_runtime.h"
#include "shfl_array_match.h"

__global__ void shfl_array_match(int* global_arrays, int num_threads) {
  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  int arr1_index = 0;
  int arr2_index = 0;

  //Stage 0: Retrieve from global memory
  for (int i = 0; i < size; i++) {
    arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }

  //Stage 1: Match by shuffle arrays
  warp_match(thread_id, current_arr1, current_arr2);

  if (num_threads > WARP_SIZE) {

    __syncthreads();

    //Stage 2: Write to shared memory
    if ((thread_id % WARP_SIZE) == 0) {

      for(int i = 0; i < size; i++) {
        arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
        shared_arrays[arr1_index] = current_arr1[i];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = ((thread_id / WARP_SIZE) * 2 * size) + size + i;
        shared_arrays[arr2_index] = current_arr2[i];
      }
    }

    __syncthreads();

    //Stage 3: Read from shared memory
    if (thread_id < WARP_SIZE) {
      for(int i = 0; i < size; i++) {
        arr1_index = (thread_id * 2 * size) + i;
        current_arr1[i] = shared_arrays[arr1_index];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = (thread_id * 2 * size) + size + i;
        current_arr2[i] = shared_arrays[arr2_index];
      }
    }

    __syncthreads();

    //Step 4: Shuffle again
    if (thread_id < WARP_SIZE) {
      warp_match(thread_id, current_arr1, current_arr2);
    }
  }

  __syncthreads();

  //Stage 5: Write back to global memory
  if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}
}
