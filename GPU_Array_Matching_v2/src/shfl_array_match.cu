#include "hip/hip_runtime.h"
#include "shfl_array_match.h"

__global__ void shfl_array_match(int* global_arrays, int num_threads) {

  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
	int next_arr1[ARRAY_SIZE];
	int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  int arr1_index = 0;
  int arr2_index = 0;
  unsigned int mask = 0xffffffff;

  //Stage 0: Retrieve from global memory
  for (int i = 0; i < size; i++) {
    arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }

  __syncthreads();

  //Stage 1: Match by shuffle arrays
  for (int delta = 1; delta < 32; delta *= 2) {

    for (int i = 0; i < size; i++) {
      next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
      next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
    }

    __syncthreads();

    if ((thread_id % (delta * 2)) == 0) {
      match(current_arr2, next_arr1, next_arr2);
    }

    __syncthreads();
  }

  //Stage 2: Write to shared memory
  if ((thread_id % WARP_SIZE) == 0) {
    for(int i = 0; i < size; i++) {
      arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
      shared_arrays[arr1_index] = current_arr1[i];

      arr2_index = ((thread_id / WARP_SIZE) * 2 * size) + size + i;
      shared_arrays[arr2_index] = current_arr2[i];
    }
  }

  __syncthreads();

  //Stage 3: Reload memory
  if (thread_id < WARP_SIZE) {
    for(int i = 0; i < size; i++) {
      arr1_index = (thread_id * 2 * size) + i;
      arr2_index = (thread_id * 2 * size) + size + i;

      current_arr1[i] = shared_arrays[arr1_index];
      current_arr2[i] = shared_arrays[arr2_index];
    }
  }

  __syncthreads();

  //Step 4: Shuffle again
  for (int delta = 1; delta < 32; delta *= 2) {

    if (thread_id < WARP_SIZE) {
      for (int i = 0; i < size; i++) {
        next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
        next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
      }
    }

    __syncthreads();

    if (thread_id < WARP_SIZE) {
      if ((thread_id % (delta * 2)) == 0) {
        match(current_arr2, next_arr1, next_arr2);
      }
    }

    __syncthreads();
  }

  //Stage 5: Write back to global memory
  if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}
}
