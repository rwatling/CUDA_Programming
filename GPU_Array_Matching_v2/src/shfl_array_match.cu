#include "hip/hip_runtime.h"
#include "shfl_array_match.h"

__global__ void shfl_array_match(int* global_arrays, int num_threads) {

  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	extern __shared__ int shared_arrays[];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
	int next_arr1[ARRAY_SIZE];
	int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  unsigned int mask = 0xffffffff;

  //Stage 0: Retrieve from global memory
  for (int i = 0; i < size; i++) {
    int arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    int arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }

  __syncthreads();

  //Stage 1: Match by shuffle arrays
  for (int delta = 1; delta < 32; delta *= 2) {

    for (int i = 0; i < size; i++) {
      next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
      next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
    }

    __syncthreads();

    if ((thread_id % (delta * 2)) == 0) {
      match(current_arr2, next_arr1, next_arr2);
    }

    __syncthreads();
  }

  //Stage 2: Write back to shared memory
  if ((thread_id % WARP_SIZE) == 0) {
    for(int i = 0; i < size; i++) {
      int arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
      shared_arrays[arr1_index] = current_arr1[i];
    }

    for(int i = 0; i < size; i++) {
      int arr2_index = (thread_id / WARP_SIZE * 2 * size) + size + i;
      shared_arrays[arr2_index] = current_arr2[i];
    }
  }

  __syncthreads();

  //Stage 3: Reload memory and shuffle again
  /*if (thread_id < WARP_SIZE) {
    //Reloard memory
    for (int i = 0; i < size; i++) {
  		int arr1_index = (thread_id * 2 * size) + i;
  		current_arr1[i] = shared_arrays[arr1_index];

  		int arr2_index = (thread_id * 2 * size) + size + i;
  		current_arr2[i] = shared_arrays[arr2_index];
  	}

      for (int delta = 1; delta < 32; delta *= 2) {

      for (int i = 0; i < size; i++) {
        next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
        next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
      }

      __syncthreads();

      if ((thread_id % (delta * 2)) == 0) {
        match(current_arr2, next_arr1, next_arr2);
      }

      __syncthreads();
    }
  }*/

  //Stage 4:Write back to global memory
  if (thread_id == 0) {
		for (int i = 0; i < 2 * size; i++) {
			int arr_index = (thread_id * 2 * size) + i;
			global_arrays[arr_index] = shared_arrays[arr_index];
		}
	}
}
