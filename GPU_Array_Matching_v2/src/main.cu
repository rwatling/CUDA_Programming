#include "hip/hip_runtime.h"
/* This program searches "states" for matches in their arrays using CUDA
*
* Author: Robbie Watling
*/

#include "cuda_includes.h"
#include "shm_array_match.h"
#include "shfl_array_match.h"
#include "cpu_array_match.h"
#include <iostream>
#include <sys/time.h>

#define SHM_96_KB 98304
#define SHM_64_KB 65536

using namespace std;

// For shuffling host arrays
void shuffle(int *array, size_t n)
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  long int mytime = tp.tv_sec * 1000 + tp.tv_usec;
  srand(mytime);

  if (n > 1) {
      int i;
      for (i = 0; i < n - 1; i++){
        int j = i + rand() / (RAND_MAX / (n - i) + 1);
        int t = array[j];
        array[j] = array[i];
        array[i] = t;
      }
  }
}

int main(int argc, char** argv) {

	/***Variable Declarations***/
  float milliseconds;

  int* host_arrays;
  int* experiment1_arrays;
  int* experiment2_arrays;
	int* device_arrays;

	int array_size;
  int num_arrays;
	int num_threads;
	int num_blocks;
  int share_size;
  int debug;

	size_t one_t;
	size_t array_set_bytes;

  hipEvent_t start, stop;
  hipEvent_t start1, stop1;
  hipError_t cuda_err;

	/*** Read args ***/
	if (argc < 3) {
		cerr << "./gpu_match num_operating_threads debug(1 or 0)" << endl;
		return -1;
	}

	/***Initialization***/
	array_size = ARRAY_SIZE;
	num_arrays = atoi(argv[1]);
  debug = (atoi(argv[2]));
  num_threads = num_arrays;
	num_blocks = 1;
  share_size = SHM_64_KB;


	//Host allocation
	one_t = (size_t) 1;
	array_set_bytes = (size_t) num_threads * array_size * 2 * sizeof(int);
  host_arrays = (int*) calloc(one_t, array_set_bytes);

	if (host_arrays == NULL) {
		cerr << "Host arrays calloc failed\n" << endl;
		return -1;
	}

  //Experiment arrays allocation
  experiment1_arrays = (int*) calloc(one_t, array_set_bytes);

  if (experiment1_arrays == NULL) {
		cerr << "experiment1 arrays calloc failed\n" << endl;
		return -1;
	}

  experiment2_arrays = (int*) calloc(one_t, array_set_bytes);

  if (experiment2_arrays == NULL) {
		cerr << "experiment2 arrays calloc failed\n" << endl;
		return -1;
	}

	//Device Allocation
	cuda_err = hipMalloc((void**)&device_arrays, array_set_bytes);

	if (cuda_err != hipSuccess) {
		cerr << "Device allocation for array set failed" << endl;
		return -1;
	}

  //Fill in host arrays to emulate major operation
  for(int i = 0; i < num_threads; i++) {

    //Start array
		for(int j = 0; j < array_size; j++) {
      if (i != 0) {
        host_arrays[(i * array_size * 2) + j] = j;
      }
		}

    if (i != 0) { shuffle(host_arrays + (i * array_size * 2), array_size); }

    //End array
    for(int j = array_size; j < array_size * 2; j++) {
      host_arrays[(i * array_size * 2) + j] = j % array_size;
		}

    shuffle(host_arrays + (i * array_size * 2) + array_size, array_size);
	}

  //Print arrays before matching
  if (debug) {
    for(int i = 0; i < num_threads; i++) {

      cout << "Arrays " << i << ": [";

  		for(int j = 0; j < array_size * 2; j++) {
  			cout << host_arrays[(i * array_size * 2) + j] << " ";

        if (j == array_size - 1) { cout << "]\t["; }
  		}

      cout << "]" << endl;
  	}
  }

  /************************Experiment 1***************************************/

  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  share_size = SHM_96_KB;
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shm_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {

    cerr << endl << "Dynamic shared memory size of 96kb for array set failed, trying 64kb" << endl;
    share_size = SHM_64_KB;

    cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shm_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

    if (cuda_err != hipSuccess) {
      cerr << "Dynamic shared memory size of 64000 for array set failed. Exiting program..." << endl;

      return -1;
    }
	}

  cout << endl << "***Experiment1***" << endl;

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  cout << "--------------------KERNEL CALL--------------------" << endl;

  //Timing
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //Kernel call
  shm_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Copy device arrays back to host
  hipMemcpy(experiment1_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  //Print arrays after matching
  for(int i = 0; i < 1; i++) {

    cout << "Arrays " << i << ": [";

    for(int j = 0; j < array_size * 2; j++) {
      cout << experiment1_arrays[(i * array_size * 2) + j] << " ";

      if (j == array_size - 1) { cout << "]\t["; }
    }

    cout << "]" << endl;
  }

  cout << milliseconds << "ms" << endl << endl;

  /************************Experiment 2***************************************/
  //Set max dynamic shared memory size to either 96 kibibytes or 64 kibibytes
  cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(shfl_array_match), hipFuncAttributeMaxDynamicSharedMemorySize, share_size);

  if (cuda_err != hipSuccess) {
    cerr << endl << "Second attempt of defining dynamic shared memory size of 96kb for array set failed" << endl << endl;
    return -1;
	}

  //Copy host arrays to device
  hipMemcpy(device_arrays, host_arrays, array_set_bytes, hipMemcpyHostToDevice);

  cout << endl << "***Experiment2***" << endl;

  cout << "--------------------KERNEL CALL--------------------" << endl;

  //Timing
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventRecord(start1, 0);

  //Kernel call
  shfl_array_match <<<num_blocks, num_threads, share_size>>> (device_arrays, num_threads);

  //Timing
  hipEventRecord(stop1, 0);
  hipEventSynchronize(stop1);
  hipEventElapsedTime(&milliseconds, start1, stop1);
  hipEventDestroy(start1);
  hipEventDestroy(stop1);

  //Copy device arrays back to host
  hipMemcpy(experiment2_arrays, device_arrays, array_set_bytes, hipMemcpyDeviceToHost);

  //Print arrays after matching
  for(int i = 0; i < 1; i++) {

    cout << "Arrays " << i << ": [";

    for(int j = 0; j < array_size * 2; j++) {
      cout << experiment2_arrays[(i * array_size * 2) + j] << " ";

      if (j == array_size - 1) { cout << "]\t["; }
    }

    cout << "]" << endl;
  }

  cout << milliseconds << "ms" << endl;

  cout << endl << "***Host Arrays***" << endl;

  cpu_array_match(host_arrays, num_threads, array_size);

  for(int i = 0; i < 1; i++) {

    cout << "Arrays " << i << ": [";

    for(int j = 0; j < array_size * 2; j++) {
      cout << host_arrays[(i * array_size * 2) + j] << " ";

      if (j == array_size - 1) { cout << "]\t["; }
    }

    cout << "]" << endl;
  }

	/***Free variables***/
	hipFree(device_arrays);
	free(host_arrays);
  free(experiment1_arrays);
  free(experiment2_arrays);

	return 0;
}
