#include "hip/hip_runtime.h"
#include "shfl_hash_w_shared_match.h"

__global__ void shfl_hash_w_shared_match(int* global_arrays, int num_threads) {
  int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
  __shared__ int shared_arrays[ARRAY_SIZE * WARP_SIZE];
  __shared__ int h_table1[HASH_SIZE][MAX_COLLISIONS];
  __shared__ int h_table2[HASH_SIZE][MAX_COLLISIONS];
	int current_arr1[ARRAY_SIZE];
	int current_arr2[ARRAY_SIZE];
  int next_arr1[ARRAY_SIZE];
  int next_arr2[ARRAY_SIZE];
	int size = ARRAY_SIZE;
  int arr1_index = 0;
  int arr2_index = 0;
  int key = 0;
  int hashed_key = 0;
  unsigned int mask = 0xffffffff;

  //Stage 0: Retrieve from global memory
  for (int i = 0; i < size; i++) {
    arr1_index = (thread_id * 2 * size) + i;
    current_arr1[i] = global_arrays[arr1_index];

    arr2_index = (thread_id * 2 * size) + size + i;
    current_arr2[i] = global_arrays[arr2_index];
  }

  //Stage 1: Match by shuffle arrays with tree like reduction
  for (int delta = 1; delta < WARP_SIZE; delta = delta << 1) {

    //Retrieve value from register from thread_id + delta
    for (int i = 0; i < size; i++) {
      next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
      next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
    }

    if ((thread_id % (delta * 2)) == 0) {
      hash_match(current_arr2, next_arr1, next_arr2);
    }
  }

  if (num_threads > WARP_SIZE) {

    //Stage 2: Warp thread 0 write warp shuffle result to shared memory
    if ((thread_id % WARP_SIZE) == 0) {

      for(int i = 0; i < size; i++) {
        arr1_index = ((thread_id / WARP_SIZE) * 2 * size) + i;
        shared_arrays[arr1_index] = current_arr1[i];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = ((thread_id / WARP_SIZE) * 2 * size) + size + i;
        shared_arrays[arr2_index] = current_arr2[i];
      }
    }

    __syncthreads();

    //Stage 3: Read all warps 0 thread from shared memory
    if (thread_id < WARP_SIZE) {
      for(int i = 0; i < size; i++) {
        arr1_index = (thread_id * 2 * size) + i;
        current_arr1[i] = shared_arrays[arr1_index];
      }

      for(int i = 0; i < size; i++) {
        arr2_index = (thread_id * 2 * size) + size + i;
        current_arr2[i] = shared_arrays[arr2_index];
      }
    }

    __syncthreads();

    //Stage 4: Shuffle again
    if (thread_id < WARP_SIZE) {

      // Tree like reduction, notice for loop condition
      for (int delta = 1; delta < (num_threads / WARP_SIZE); delta = delta << 1) {

        //Retrieve value from register from thread_id + delta
        for (int i = 0; i < size; i++) {
          next_arr1[i] = __shfl_down_sync(mask, current_arr1[i], delta, WARP_SIZE);
          next_arr2[i] = __shfl_down_sync(mask, current_arr2[i], delta, WARP_SIZE);
        }

        if ((thread_id % (delta * 2)) == 0) {
          //Hash match
          //Hash tables are not garunteed to be 0
          for (int i = 0; i < HASH_SIZE; i++) {
            for (int j = 0; j < MAX_COLLISIONS; j++) {
              h_table1[i][j] = 0;
            }
          }

          //Hash "next" arrays
          for (int i = 0; i < ARRAY_SIZE; i++) {
            key = next_arr1[i];
            hashed_key = hash(key);

            if (h_table1[hashed_key][0] == 0) {
              h_table1[hashed_key][0] = next_arr1[i];
              h_table2[hashed_key][0] = next_arr2[i];
            } else if (h_table1[hashed_key][1] == 0) {
              h_table1[hashed_key][1] = next_arr1[i];
              h_table2[hashed_key][1] = next_arr2[i];
            }
          }

          //Find values
          for (int i = 0; i < ARRAY_SIZE; i++) {
            key = current_arr2[i];
            hashed_key = hash(key);

            //array2[i] = h_table2[hashed_key][0];

            if (key == h_table1[hashed_key][0]) {
              current_arr2[i] = h_table2[hashed_key][0];
            } else if (key == h_table1[hashed_key][1]) {
              current_arr2[i] = h_table2[hashed_key][1];
            }
          }
        }
      }
    }
  }

  //Stage 5: Write back to global memory
  if (thread_id == 0) {
		for (int i = 0; i < size; i++) {
			arr1_index = (thread_id * 2 * size) + i;
			global_arrays[arr1_index] = current_arr1[i];

			arr2_index = (thread_id * 2 * size) + size + i;
			global_arrays[arr2_index] = current_arr2[i];
		}
	}
}
